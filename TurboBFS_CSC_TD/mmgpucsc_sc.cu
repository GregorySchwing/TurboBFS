#include "hip/hip_runtime.h"
/*
 * This source code is distributed under the terms defined  
 * in the file bfstdcsc_main.c of this source distribution.
 */
/* 
 *  Breadth first search (BFS) 
 *  Single precision (float data type) 
 *  TurboBFS_CSC_TD:bfsgputdcsc_sc.cu
 * 
 *  This program computes the GPU-based parallel 
 *  top-down BFS (scalar) for unweighted graphs represented 
 *  by sparse adjacency matrices in the CSC format, including
 *  the computation of the S array to store the depth at 
 *  which each vertex is discovered.  
 *
 */

#include <cstdlib>
#include <iostream>
#include <cassert>
#include <cmath>

//includes CUDA project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "bfsgputdcsc.cuh"
extern "C"{
                 #include "bfstdcsc.h"

}


/*************************prototype kernel*************************************/
__global__ void spMvUgCscScKernel (int *CP_d,int *IC_d,int *ft_d,int *f_d,
				   float *sigma_d,int j,int r,int n);
/******************************************************************************/

/* 
 * Function to compute a gpu-based parallel top-down BFS (scalar) for 
 * unweighted graphs represented by sparse adjacency matrices in CSC format,
 * including the computation of the S vector to store the depth at which each  
 * vertex is  discovered.
 *  
 */
int  bfs_gpu_mm_csc_sc (int *IC_h,int *CP_h,int *m_h,int nz,int n,int repetition){
  float t_spmv;
  float t_spmv_t = 0.0;
  float t_bfsfunctions;
  float t_bfsfunctions_t = 0.0;
  float t_sum = 0.0;
  float t_bfs_avg;
  int i,d = 0,dimGrid;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /*Allocate device memory for the vector CP_d */
  int *CP_d;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&CP_d),sizeof(*CP_d)*(n+1)));
  /*Copy host memory (CP_h) to device memory (CP_d)*/
  checkCudaErrors(hipMemcpy(CP_d,CP_h,(n+1)*sizeof(*CP_d),hipMemcpyHostToDevice));

  /*Allocate device memory for the vector IC_d */
  int *IC_d;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&IC_d),sizeof(*IC_d)*nz));
  /*Copy host memory (IC_h) to device memory (IC_d)*/
  checkCudaErrors(hipMemcpy(IC_d,IC_h,nz*sizeof(*IC_d),hipMemcpyHostToDevice));

  /*Allocate device memory for the vector m_d, and set m_d to zero. */
  int *m_d;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&m_d),sizeof(*m_d)*n));
  checkCudaErrors(hipMemset(m_d,0,sizeof(*m_d)*n));

  /*Allocate device memory for the vector f_d*/
  int *req_d;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&req_d),sizeof(*req_d)*n));

  /*allocate unified memory for integer variable c for control of while loop*/
  int *c;
  checkCudaErrors(hipMallocManaged(reinterpret_cast<void **>(&c),sizeof(*c)));

  /*computing BFS */
  dimGrid = (n + THREADS_PER_BLOCK)/THREADS_PER_BLOCK;
  for (i = 0; i<repetition; i++){
    *c = 1;
    d = 0;
    checkCudaErrors(hipMemset(req_d,0,sizeof(*req_d)*n));
    while (*c){
      d = d + 1;
      *c = 0;

      hipEventRecord(start);
      //spMvUgCscScKernel <<<dimGrid,THREADS_PER_BLOCK>>> (CP_d,IC_d,ft_d,f_d,sigma_d,d,r,n);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&t_spmv,start,stop);
      t_spmv_t += t_spmv;

      hipEventRecord(start);
      //bfsFunctionsKernel <<<dimGrid,THREADS_PER_BLOCK>>> (f_d,ft_d,sigma_d,m_d,c,n,d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&t_bfsfunctions,start,stop);
      t_bfsfunctions_t += t_bfsfunctions;
      
      t_sum += t_spmv + t_bfsfunctions;
    }
  }
  printf("\bfs_gpu_mm_csc_sc::t_sum=%lfms \n",t_sum);
  t_bfs_avg = t_sum/repetition;

  /*Copy device memory (m_d) to host memory (S_h)*/
  checkCudaErrors(hipMemcpy(m_h,m_d, n*sizeof(*m_d),hipMemcpyDeviceToHost));

  int print_t = 1;
  if (print_t){
    printf("bfsgputdcsc_sc::time f <-- fA d = %lfms \n",t_spmv_t/repetition);
    printf("bfsgputdcsc_sc::time time bfs functions d = %lfms \n", t_bfsfunctions_t/repetition);
    printf("bfsgputdcsc_sc::average time BFS d = %lfms \n",t_bfs_avg);
  }

  /*cleanup memory*/
  checkCudaErrors(hipFree(CP_d));
  checkCudaErrors(hipFree(IC_d));
  checkCudaErrors(hipFree(m_d));
  checkCudaErrors(hipFree(req_d));
  checkCudaErrors(hipFree(c));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  return 0;
}//end bfs_gpu_td_csc_sc